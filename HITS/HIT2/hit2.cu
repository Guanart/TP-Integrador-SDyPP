
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void holamundo_cuda() {
    int bloque_tamanio = blockDim.x;
    int bloque_numero = blockIdx.x;
    int thread_numero = threadIdx.x;
    
    int id_ejecucion = bloque_numero * bloque_tamanio + thread_numero;
    printf("Hola mundo desde ejecucion numero %i\n", id_ejecucion);
}

int main() {
    int bloque_cantidad = 1;
    int bloque_tamanio = 896;

    holamundo_cuda<<<bloque_cantidad, bloque_tamanio>>>();
    hipDeviceSynchronize();
}

